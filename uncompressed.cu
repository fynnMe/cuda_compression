
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <time.h>

// CUDA Error handler to be placed around all CUDA calls
#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

#define NUM_ITERATIONS_PER_CONFIG 3

// Dummy kernel to warm up GPU
__global__ void warmup_kernel()
{
    // Empty kernel, does nothing
}

// Grid-striding kernel for vector add
__global__ void add(uint64_t *a, uint64_t *b, uint64_t *c, int num_elements){
    for (int idx = threadIdx.x + blockIdx.x*blockDim.x;
         idx < num_elements;
         idx += blockDim.x*gridDim.x) {

        c[idx] = a[idx] + b[idx];
    }
}

uint64_t generate_random_64bit() {
    uint64_t high = (uint64_t)rand(); // Generate the high 32 bits
    uint64_t low = (uint64_t)rand();  // Generate the low 32 bits

    // Shift the high part and combine with low part
    // Use & 0x7FFFFFFFFFFFFFFF to force MSB to 0
    return ((high << 32) | low) & 0x7FFFFFFFFFFFFFFF;
}

// Print 64 bits, starting from MSB
void print_binary(uint64_t num) {
    for(int i = 63; i >= 0; i--) {
        printf("%lu", (num >> i) & 1UL);

        // Add space every 8 bits for readability
        if (i % 8 == 0) {
            printf(" ");
        }
    }
    printf("\n");
}

int main (int argc, char **argv){
    if(argc != 4) {
        printf("Usage: %s <num_elements> <block_size> <grid_size>\n", argv[0]);
        return 1;
    }

    // Enable accurate printf debugging
    setbuf(stdout, NULL);

    // Open CSV
    FILE *csv_file = fopen("uncompressed_configs.csv", "a");
    if (csv_file == NULL) {
        printf("Error opening CSV file!\n");
        return 1;
    }
    // Write CSV header if needed
    fseek(csv_file, 0, SEEK_END);
    long size = ftell(csv_file);
    if (size == 0) {
        // File is empty, write header
        fprintf(csv_file, "array_size;block_size;grid_size;runtime\n");
    }

    // Rename input
    int num_elements = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int grid_size = atoi(argv[3]);

    // Use GPU 1
    // cudaSetDevice(1);

    // Initialize host data
    uint64_t* a_host = new uint64_t[num_elements];
    uint64_t* b_host = new uint64_t[num_elements];
    uint64_t* c_host = new uint64_t[num_elements];

    // Initialize device data
    uint64_t* a_device = 0;
    uint64_t* b_device = 0;
    uint64_t* c_device = 0;

    // Allocate device data
    CUDA_CHECK  ( hipMalloc((void**) &a_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &b_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &c_device, sizeof(uint64_t)*num_elements) );

    // Declare time measurment variables
    hipEvent_t start, stop;
    float tot_time_milliseconds[NUM_ITERATIONS_PER_CONFIG];
    float avg_time_milliseconds;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Invoke dummy kernel for GPU warmup
    warmup_kernel<<<1, 1>>>();
    hipError_t error = hipGetLastError(); // Check for launch errors
    if (error != hipSuccess) {
        printf("Launch error: %s\n", hipGetErrorString(error));
        return error;
    }

    error = hipDeviceSynchronize(); // Check for execution errors
    if (error != hipSuccess) {
        printf("Execution error: %s\n", hipGetErrorString(error));
        return error;
    }

    // Invoke kernel
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        // Gerenate host data
        srand((unsigned int)time(NULL));
        for (int l = 0; l < num_elements; ++l) {
            a_host[l] = generate_random_64bit();
            b_host[l] = generate_random_64bit();
        }

        // Copy data from host to device
        CUDA_CHECK  ( hipMemcpy(   a_device,
                                    a_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );
        CUDA_CHECK  ( hipMemcpy(   b_device,
                                    b_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );

        // Call kernel
        hipEventRecord(start);
        add<<<grid_size, block_size>>>(a_device, b_device, c_device, num_elements);
        hipEventRecord(stop);
        error = hipGetLastError(); // Check for launch errors
        if (error != hipSuccess) {
            printf("Launch error: %s\n", hipGetErrorString(error));
            return error;
        }

        error = hipDeviceSynchronize(); // Check for execution errors
        if (error != hipSuccess) {
            printf("Execution error: %s\n", hipGetErrorString(error));
            return error;
        }
        hipEventSynchronize(stop); // Wait for the stop event to complete
        hipEventElapsedTime(&tot_time_milliseconds[k], start, stop);

        // Copy back result from device to host
        CUDA_CHECK  ( hipMemcpy(   c_host,
                                    c_device,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyDeviceToHost)
                    );
    }

    // Calculate average runtime
    avg_time_milliseconds = 0;
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        avg_time_milliseconds += tot_time_milliseconds[k];
    }
    avg_time_milliseconds = avg_time_milliseconds / NUM_ITERATIONS_PER_CONFIG;

    // Print average runtime
    printf("num_elements: %d, block size: %d, grid_size: %d, runtime: %.6fms\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // Add csv data entries
    fprintf(csv_file, "%d;%d;%d;%.6f\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // free memory on GPU
    CUDA_CHECK( hipFree(a_device) );
    CUDA_CHECK( hipFree(b_device) );
    CUDA_CHECK( hipFree(c_device) );

    // free memory on host
    delete[] a_host;
    delete[] b_host;
    delete[] c_host;
    
    fclose(csv_file);

    return 0;
}
