
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <time.h>

// CUDA Error handler to be placed around all CUDA calls
#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

#define NUM_ITERATIONS_PER_CONFIG 3
#define BITSIZE 11
#define ELEMENTS_PER_INT 64 / BITSIZE

// Dummy kernel to warm up GPU
__global__ void warmup_kernel()
{
    // Empty kernel, does nothing
}

// Grid-striding kernel for vector add with variable static bit size for all elements in a[i] and b[i]
__global__ void add(uint64_t *a, uint64_t *b, uint64_t *c, int num_elements) {
    for (int idx = threadIdx.x + blockIdx.x*blockDim.x;
         idx < num_elements;
         idx += blockDim.x*gridDim.x) {
        
        // Arrays based on number of elements
        int elements_per_block = blockDim.x/ELEMENTS_PER_INT;
        extern __shared__ uint64_t shared_mem[];
        uint64_t* a_block = shared_mem;
        uint64_t* b_block = &shared_mem[elements_per_block];
        uint64_t* c_block = &shared_mem[2 * elements_per_block];


        // First thread in a block initializes memory
        if (threadIdx.x == 0) {
            for (int i = 0; i < elements_per_block; ++i) {
                a_block[i] = a[i + blockIdx.x*blockDim.x];
                b_block[i] = b[i + blockIdx.x*blockDim.x];
                c_block[i] = 0;
            }
        }

        __syncthreads(); // Wait until all threads in a block reach this point

        uint64_t base_mask = (1ULL << BITSIZE) - 1;
        uint64_t position_within_uint64 = threadIdx.x % ELEMENTS_PER_INT;
        uint64_t bitmask = base_mask << (position_within_uint64 * BITSIZE);
        uint64_t a_component, b_component, c_component;
        int index_of_uint64_in_array = threadIdx.x/ELEMENTS_PER_INT;

        // Extract components
        a_component = (a_block[index_of_uint64_in_array] & bitmask) >> (position_within_uint64 * BITSIZE);
        b_component = (b_block[index_of_uint64_in_array] & bitmask) >> (position_within_uint64 * BITSIZE);

        // Perform addition
        c_component = a_component + b_component;

        // Compress
        atomicOr((unsigned long long*)&c_block[index_of_uint64_in_array], 
         (unsigned long long)(c_component << (position_within_uint64 * BITSIZE)));

        __syncthreads(); // Wait until all threads in a block reach this point
        
        // First thread in a block copies back
        if (threadIdx.x == 0) {
            for (int i = 0; i < elements_per_block; ++i) {
                c[i + blockIdx.x*blockDim.x] = c_block[i];
            }
        }
    }
}

uint64_t generate_random_64bit() {
    uint64_t high = (uint64_t)rand(); // Generate the high 32 bits
    uint64_t low = (uint64_t)rand();  // Generate the low 32 bits
    uint64_t result = (high << 32) | low;
    
    // Calculate mask for one element of given BITSIZE
    // Example: for 4-bit elements, single_element_mask = 0x0F
    uint64_t single_element_mask = (1ULL << (BITSIZE - 1)) - 1;
    
    // Calculate full mask for all elements
    uint64_t full_mask = 0;
    
    for (int i = 0; i < ELEMENTS_PER_INT; i++) {
        full_mask |= (single_element_mask << (i * BITSIZE));
    }
    
    // Apply mask to ensure MSB is 0 for each element of given BITSIZE
    return result & full_mask;
}

// Print 64 bits, starting from MSB
void print_binary(uint64_t num) {
    for(int i = 63; i >= 0; i--) {
        printf("%lu", (num >> i) & 1UL);

        // Add space every 8 bits for readability
        if (i % 8 == 0) {
            printf(" ");
        }
    }
    printf("\n");
}

int main (int argc, char **argv){
    if(argc != 4) {
        printf("Usage: %s <num_elements> <block_size> <grid_size>\n", argv[0]);
        return 1;
    }

    // Enable accurate printf debugging
    setbuf(stdout, NULL);

    // Open CSV
    FILE *csv_file = fopen("compressed_configs.csv", "a");
    if (csv_file == NULL) {
        printf("Error opening CSV file!\n");
        return 1;
    }

    // Write CSV header if needed
    fseek(csv_file, 0, SEEK_END);
    long size = ftell(csv_file);
    if (size == 0) {
        // File is empty, write header
        fprintf(csv_file, "array_size;block_size;grid_size;runtime\n");
    }

    // Rename input
    int num_elements = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int grid_size = atoi(argv[3]);

    // Use GPU 1
    // cudaSetDevice(1);

    // Initialize host data
    uint64_t* a_host = new uint64_t[num_elements];
    uint64_t* b_host = new uint64_t[num_elements];
    uint64_t* c_host = new uint64_t[num_elements];

    // Initialize device data
    uint64_t* a_device = 0;
    uint64_t* b_device = 0;
    uint64_t* c_device = 0;

    // Allocate device data
    CUDA_CHECK  ( hipMalloc((void**) &a_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &b_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &c_device, sizeof(uint64_t)*num_elements) );

    // Declare time measurment variables
    hipEvent_t start, stop;
    float tot_time_milliseconds[NUM_ITERATIONS_PER_CONFIG];
    float avg_time_milliseconds;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Invoke dummy kernel for GPU warmup
    warmup_kernel<<<1, 1>>>();
    hipError_t error = hipGetLastError(); // Check for launch errors
    if (error != hipSuccess) {
        printf("Launch error: %s\n", hipGetErrorString(error));
        return error;
    }

    error = hipDeviceSynchronize(); // Check for execution errors
    if (error != hipSuccess) {
        printf("Execution error: %s\n", hipGetErrorString(error));
        return error;
    }

    // Invoke kernel
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        // Gerenate host data
        srand((unsigned int)time(NULL));
        for (int l = 0; l < num_elements; ++l) {
            a_host[l] = generate_random_64bit();
            b_host[l] = generate_random_64bit();
        }

        // Copy data from host to device
        CUDA_CHECK  ( hipMemcpy(   a_device,
                                    a_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );
        CUDA_CHECK  ( hipMemcpy(   b_device,
                                    b_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );

        // Call kernel
        hipEventRecord(start);
        add<<<grid_size, block_size, 3*(block_size/ELEMENTS_PER_INT)>>>(a_device, b_device, c_device, num_elements);
        hipEventRecord(stop);
        error = hipGetLastError(); // Check for launch errors
        if (error != hipSuccess) {
            printf("Launch error: %s\n", hipGetErrorString(error));
            return error;
        }

        error = hipDeviceSynchronize(); // Check for execution errors
        if (error != hipSuccess) {
            printf("Execution error: %s\n", hipGetErrorString(error));
            return error;
        }
        hipEventSynchronize(stop); // Wait for the stop event to complete
        hipEventElapsedTime(&tot_time_milliseconds[k], start, stop);

        // Copy back result from device to host
        CUDA_CHECK  ( hipMemcpy(   c_host,
                                    c_device,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyDeviceToHost)
                    );
    }

    // Calculate average runtime
    avg_time_milliseconds = 0;
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        avg_time_milliseconds += tot_time_milliseconds[k];
    }
    avg_time_milliseconds = avg_time_milliseconds / NUM_ITERATIONS_PER_CONFIG;

    // Print average runtime
    printf("num_elements: %d, block size: %d, grid_size: %d, runtime: %.6fms\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // Add csv data entries
    fprintf(csv_file, "%d;%d;%d;%.6f\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // free memory on GPU
    CUDA_CHECK( hipFree(a_device) );
    CUDA_CHECK( hipFree(b_device) );
    CUDA_CHECK( hipFree(c_device) );

    // free memory on host
    delete[] a_host;
    delete[] b_host;
    delete[] c_host;

    fclose(csv_file);

    return 0;
}
