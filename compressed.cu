
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <time.h>

// CUDA Error handler to be placed around all CUDA calls
#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

#define NUM_ITERATIONS_PER_CONFIG 3
#define BITSIZE 11
#define ELEMENTS_PER_INT 64 / BITSIZE

// Dummy kernel to warm up GPU
__global__ void warmup_kernel()
{
    // Empty kernel, does nothing
}

// Grid-striding kernel for vector add with variable static bit size for all elements in a[i] and b[i]
__global__ void add(uint64_t *a, uint64_t *b, uint64_t *c, int num_elements) {
    for (int idx = threadIdx.x + blockIdx.x*blockDim.x;
         idx < num_elements;
         idx += blockDim.x*gridDim.x) {
        
        // Create initial bitmask for one element
        uint64_t single_element_mask = (1ULL << BITSIZE) - 1;
        uint64_t bitmask = single_element_mask;
        
        // Dynamic arrays based on number of elements
        uint64_t a_components[ELEMENTS_PER_INT];  // Max possible size
        uint64_t b_components[ELEMENTS_PER_INT];
        uint64_t c_components[ELEMENTS_PER_INT];

        c[idx] = 0;

        for (int i = 0; i < ELEMENTS_PER_INT; ++i) {
            // Extract components
            a_components[i] = (a[idx] & bitmask) >> (i * BITSIZE);
            b_components[i] = (b[idx] & bitmask) >> (i * BITSIZE);
            bitmask = bitmask << BITSIZE;

            // Perform addition
            c_components[i] = a_components[i] + b_components[i];

            // Compress
            c[idx] = c[idx] | (c_components[i] << (i * BITSIZE));
        }
    }
}

uint64_t generate_random_64bit() {
    uint64_t high = (uint64_t)rand(); // Generate the high 32 bits
    uint64_t low = (uint64_t)rand();  // Generate the low 32 bits
    uint64_t result = (high << 32) | low;
    
    // Calculate mask for one element of given BITSIZE
    // Example: for 4-bit elements, single_element_mask = 0x0F
    uint64_t single_element_mask = (1ULL << (BITSIZE - 1)) - 1;
    
    // Calculate full mask for all elements
    uint64_t full_mask = 0;
    
    for (int i = 0; i < ELEMENTS_PER_INT; i++) {
        full_mask |= (single_element_mask << (i * BITSIZE));
    }
    
    // Apply mask to ensure MSB is 0 for each element of given BITSIZE
    return result & full_mask;
}

// Print 64 bits, starting from MSB
void print_binary(uint64_t num) {
    for(int i = 63; i >= 0; i--) {
        printf("%lu", (num >> i) & 1UL);

        // Add space every 8 bits for readability
        if (i % 8 == 0) {
            printf(" ");
        }
    }
    printf("\n");
}

int main (int argc, char **argv){
    if(argc != 4) {
        printf("Usage: %s <num_elements> <block_size> <grid_size>\n", argv[0]);
        return 1;
    }

    // Enable accurate printf debugging
    setbuf(stdout, NULL);

    // Open CSV
    FILE *csv_file = fopen("compressed_configs.csv", "a");
    if (csv_file == NULL) {
        printf("Error opening CSV file!\n");
        return 1;
    }

    // Write CSV header if needed
    fseek(csv_file, 0, SEEK_END);
    long size = ftell(csv_file);
    if (size == 0) {
        // File is empty, write header
        fprintf(csv_file, "array_size;block_size;grid_size;runtime\n");
    }

    // Rename input
    int num_elements = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int grid_size = atoi(argv[3]);

    // Use GPU 1
    // cudaSetDevice(1);

    // Initialize host data
    uint64_t* a_host = new uint64_t[num_elements];
    uint64_t* b_host = new uint64_t[num_elements];
    uint64_t* c_host = new uint64_t[num_elements];

    // Initialize device data
    uint64_t* a_device = 0;
    uint64_t* b_device = 0;
    uint64_t* c_device = 0;

    // Allocate device data
    CUDA_CHECK  ( hipMalloc((void**) &a_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &b_device, sizeof(uint64_t)*num_elements) );
    CUDA_CHECK  ( hipMalloc((void**) &c_device, sizeof(uint64_t)*num_elements) );

    // Declare time measurment variables
    hipEvent_t start, stop;
    float tot_time_milliseconds[NUM_ITERATIONS_PER_CONFIG];
    float avg_time_milliseconds;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Invoke dummy kernel for GPU warmup
    warmup_kernel<<<1, 1>>>();
    hipError_t error = hipGetLastError(); // Check for launch errors
    if (error != hipSuccess) {
        printf("Launch error: %s\n", hipGetErrorString(error));
        return error;
    }

    error = hipDeviceSynchronize(); // Check for execution errors
    if (error != hipSuccess) {
        printf("Execution error: %s\n", hipGetErrorString(error));
        return error;
    }

    // Invoke kernel
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        // Gerenate host data
        srand((unsigned int)time(NULL));
        for (int l = 0; l < num_elements; ++l) {
            a_host[l] = generate_random_64bit();
            b_host[l] = generate_random_64bit();
        }

        // Copy data from host to device
        CUDA_CHECK  ( hipMemcpy(   a_device,
                                    a_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );
        CUDA_CHECK  ( hipMemcpy(   b_device,
                                    b_host,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyHostToDevice)
                    );

        // Call kernel
        hipEventRecord(start);
        add<<<grid_size, block_size>>>(a_device, b_device, c_device, num_elements);
        hipEventRecord(stop);
        error = hipGetLastError(); // Check for launch errors
        if (error != hipSuccess) {
            printf("Launch error: %s\n", hipGetErrorString(error));
            return error;
        }

        error = hipDeviceSynchronize(); // Check for execution errors
        if (error != hipSuccess) {
            printf("Execution error: %s\n", hipGetErrorString(error));
            return error;
        }
        hipEventSynchronize(stop); // Wait for the stop event to complete
        hipEventElapsedTime(&tot_time_milliseconds[k], start, stop);

        // Copy back result from device to host
        CUDA_CHECK  ( hipMemcpy(   c_host,
                                    c_device,
                                    sizeof(uint64_t)*num_elements,
                                    hipMemcpyDeviceToHost)
                    );
    }

    // Calculate average runtime
    avg_time_milliseconds = 0;
    for (int k = 0; k < NUM_ITERATIONS_PER_CONFIG; ++k) {
        avg_time_milliseconds += tot_time_milliseconds[k];
    }
    avg_time_milliseconds = avg_time_milliseconds / NUM_ITERATIONS_PER_CONFIG;

    // Print average runtime
    printf("num_elements: %d, block size: %d, grid_size: %d, runtime: %.6fms\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // Add csv data entries
    fprintf(csv_file, "%d;%d;%d;%.6f\n", num_elements, block_size, grid_size, avg_time_milliseconds);

    // free memory on GPU
    CUDA_CHECK( hipFree(a_device) );
    CUDA_CHECK( hipFree(b_device) );
    CUDA_CHECK( hipFree(c_device) );

    // free memory on host
    delete[] a_host;
    delete[] b_host;
    delete[] c_host;

    fclose(csv_file);

    return 0;
}
